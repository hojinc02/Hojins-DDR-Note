
#include <hip/hip_runtime.h>
__device__ void swap(float& a, float& b) {
  float t = a;
  a = b;
  b = t;
}

__device__ bool raycast(
  float x1, 
  float y1, 
  float x2, 
  float y2,
  float a, 
  float b
) { // return true if horzontal ray shot from (a,b) to the right
    // casts on to edge (x1,y1)<->(x2,y2) 
  if (fabsf(y1 - y2) < 1e-6f) {
    return false;
  }

  if (y2 < y1) {
    swap(y1,y2);
    swap(x1,x2);
  }

  if ((y1 <= b) && (b < y2)) { // within y-range

    // find x_intercept at y=b on polygon line
    /*
    y-y1 = m (x-x1)
    b-y1 = (y2-y1)/(x2-x1) * (xi-x1)
    (b-y1)*(x2-x1)/(y2-y1) = xi-x1
    xi = x1 + (b-y1)*(x2-x1)/(y2-y1)
    */
    float x_intercept = x1 + (b - y1) * (x2 - x1) / (y2 - y1);
    if (a < x_intercept) { 
      return true;
    }
  }

  return false;
}

__device__ bool withinPolygon(
  const int a, 
  const int b, 
  const float* const polygon_vertices, 
  const int polygon_edge_count
) {
  unsigned int count = 0;

  for (int e = 0; e < polygon_edge_count-1; e+=1) {
    float x1 = polygon_vertices[2*e];
    float y1 = polygon_vertices[2*e+1];
    float x2 = polygon_vertices[2*(e+1)];
    float y2 = polygon_vertices[2*(e+1)+1];
    
    if (raycast(x1,y1,x2,y2,a,b)) {
      count += 1; 
    }
  }
  // check last edge
  float x2 = polygon_vertices[0];
  float y2 = polygon_vertices[1];
  float x1 = polygon_vertices[2*(polygon_edge_count-1)];
  float y1 = polygon_vertices[2*(polygon_edge_count-1)+1];
  if (raycast(x1,y1,x2,y2,a,b)) {
    count += 1;
  }

  if (count % 2 == 1) { // point inside polygon norm
    return true;
  } else {
    return false;
  }
}

// for every pixel, if pixel colored, color around in polygon shape
extern "C" __global__ void morph(
  const unsigned char* const in_mask, 
  unsigned char* const out_mask, 
  const long long H, 
  const long long W,
  const long long fill,
  const float* const polygon_vertices, 
  const long long polygon_edge_count
) {

  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  if (tx >= W || tx < 0 || ty >= H || ty < 0 || in_mask[ty*W+tx] == 0) {
    // skip empty pixels
    return;
  }

  float minX = 1e30f;
  float maxX = -1e30f;
  float minY = 1e30f;
  float maxY = -1e30f;
  for (int i = 0; i < polygon_edge_count * 2; i += 2) {
    if (minX > polygon_vertices[i]) minX = polygon_vertices[i];
    if (maxX < polygon_vertices[i]) maxX = polygon_vertices[i];
  }
  for (int j = 1; j < polygon_edge_count * 2; j += 2) {
    if (minY > polygon_vertices[j]) minY = polygon_vertices[j];
    if (maxY < polygon_vertices[j]) maxY = polygon_vertices[j];
  }
  int iminX = static_cast<int>(floorf(minX));
  int imaxX = static_cast<int>(ceilf(maxX));
  int iminY = static_cast<int>(floorf(minY));
  int imaxY = static_cast<int>(ceilf(maxY));

  for (int i = iminX; i <= imaxX; i++) {
  for (int j = iminY; j <= imaxY; j++) {
    int sx = tx + i; 
    int sy = ty + j;
    if (sx >= 0 && sx < W && sy >= 0 && sy < H) {
      if (withinPolygon(i, j, polygon_vertices, polygon_edge_count)) {
        out_mask[sy*W+sx] = fill;
      }
    }
  }
  }
}